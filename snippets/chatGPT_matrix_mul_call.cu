#include <hip/hip_runtime.h>
#include <stdio.h>

#include "chatGPT_matrix_mul.cu"

#define ROW_SIZE 3
#define COL_SIZE 3

void print_matrix(float* M, size_t row_size=ROW_SIZE, size_t col_size=COL_SIZE) {
    for (size_t i = 0; i < row_size; i++)
    {
        for (size_t j = 0; j < col_size-1; j++)
        {
            printf("%.2f, ", M[i*col_size+j]);
        }
        printf("%.2f\n", M[(i+1)*col_size-1]);
    }
}


int main(int argc, char **argv) {

    float *A_cpu, *B_cpu, *C_cpu;

    hipHostMalloc(&A_cpu, ROW_SIZE*COL_SIZE*sizeof(float));
    hipHostMalloc(&B_cpu, ROW_SIZE*COL_SIZE*sizeof(float));
    hipHostMalloc(&C_cpu, ROW_SIZE*COL_SIZE*sizeof(float));


    if (!A_cpu || !B_cpu || !C_cpu) {
        if (A_cpu) {
            free(A_cpu);
        }

        if (B_cpu) {
            free(B_cpu);
        }

        if (C_cpu) {
            free(C_cpu);
        }

        printf("Allocating memory failed\n");
        return -1;
    }


    float* A_gpu;
    float* B_gpu;
    float* C_gpu;

    if (hipMalloc<float>(&A_gpu, sizeof(float)*ROW_SIZE*COL_SIZE) != hipSuccess) {
        hipFree(A_cpu);
        hipFree(B_cpu);
        hipFree(C_cpu);
        printf("hipMalloc failed\n");
        return -1;
    } else if(hipMalloc<float>(&B_gpu, sizeof(float)*ROW_SIZE*COL_SIZE) != hipSuccess) {
        hipFree(A_gpu);
        hipFree(A_cpu);
        hipFree(B_cpu);
        hipFree(C_cpu);
        printf("hipMalloc failed\n");
        return -1;
    } else if (hipMalloc<float>(&C_gpu, sizeof(float)*ROW_SIZE*COL_SIZE) != hipSuccess) {
        hipFree(A_gpu);
        hipFree(B_gpu);
        hipFree(A_cpu);
        hipFree(B_cpu);
        hipFree(C_cpu);
        printf("hipMalloc failed\n");
        return -1;
    }

    for (size_t i = 0; i < ROW_SIZE; i++)
    {
        for (size_t j = 0; j < COL_SIZE; j++)
        {
            A_cpu[i*COL_SIZE + j] = i*COL_SIZE + j + 1;
            B_cpu[i*COL_SIZE + j] = i*COL_SIZE + j + 1;
        }
    }

    hipDeviceSynchronize();

    print_matrix(A_cpu);
    printf("\n");
    print_matrix(B_cpu);

    hipMemcpy(A_gpu, A_cpu, ROW_SIZE*COL_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B_cpu, ROW_SIZE*COL_SIZE, hipMemcpyHostToDevice);
    matrixMultiplication<<<64, 128>>>(A_gpu, B_gpu, C_gpu, ROW_SIZE, COL_SIZE, ROW_SIZE, COL_SIZE, ROW_SIZE, COL_SIZE);
    hipMemcpy(C_cpu, C_gpu, ROW_SIZE*COL_SIZE, hipMemcpyDeviceToHost);
    
    printf("===============================\n");
    print_matrix(C_cpu);

    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(C_gpu);
    hipFree(A_cpu);
    hipFree(B_cpu);
    hipFree(C_cpu);

    return 0;
}